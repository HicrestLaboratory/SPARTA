#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <unordered_map>

//CUDA Utilities and system includes
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

#include <cutlass/layout/matrix.h>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/kernel/gemm_grouped.h"
#include "cutlass/gemm/kernel/default_gemm_grouped.h"
#include "cutlass/gemm/device/ell_gemm.h"

#include "cutlass/util/tensor_view_io.h"
// #include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm.h"

// --------------
#include "cutlass/gemm/device/gemm_sparse.h"
#include "cutlass/util/host_reorder.h"
#include "cutlass/util/host_uncompress.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "helper.h"

#include <cutlass/numeric_types.h>
#include <cutlass/gemm/device/gemm.h>

#include <cutlass/util/host_tensor.h>
// --------------

#include "cuda_utilities.h"
#include "cutlass_bellpack_lib.h"


#define DataT float
#define DataT_Cutlass cutlass::half_t
#define NOELL_TEST

template<typename iT, typename T>
int compute_cutlass_bellpack (int rows, int cols, int ell_blocksize, int ellValue_cols, iT* ellColInd, T *ellValues, int B_rows, int B_cols, T *B_vals, int C_rows, int C_cols, T *C_vals, float& dt) {

    int i, j, ellColInd_rows = (rows/ell_blocksize), ellColInd_cols = (ellValue_cols/ell_blocksize);

//     printf("Inside of %s\n", __func__);
//     print_bellpack("Bellpack_A", rows, cols, ell_blocksize, ellValue_cols, ellColInd_rows, ellColInd_cols, ellColInd_rows*ellColInd_cols, ellColInd, ellValues);

    cutlass::HostTensor<DataT_Cutlass, cutlass::layout::RowMajor> tensor({ellColInd_rows*ell_blocksize, ellColInd_cols*ell_blocksize});
    for (i = 0; i < (ellColInd_rows*ell_blocksize); ++i) {
        for (j = 0; j < (ellColInd_cols*ell_blocksize); ++j) {
            // Write the element at location {i, j} in host memory
            tensor.host_ref().at({i, j}) = (DataT_Cutlass)ellValues[i*ellValue_cols + j];
        }
    }
    // Copy host memory to device memory
    tensor.sync_device();
    // Obtain a device pointer usable in CUDA kernels
    DataT_Cutlass *device_ptr = tensor.device_data();


    cutlass::HostTensor<int32_t, cutlass::layout::RowMajor> tensor_ellIdx({ellColInd_rows, ellColInd_cols});
    for (i = 0; i < ellColInd_rows; ++i) {
        for (j = 0; j < ellColInd_cols; ++j) {
            // Write the element at location {i, j} in host memory
            tensor_ellIdx.host_ref().at({i, j}) = (int32_t)ellColInd[i*ellColInd_cols + j];
        }
    }
    // Copy host memory to device memory
    tensor_ellIdx.sync_device();
    // Obtain a device pointer usable in CUDA kernels
    int32_t *device_ptr_ellIdx = tensor_ellIdx.device_data();

//     printf("Inside of %s\n", __func__);
//     std::cout << "view of tensor_ellIdx:" << std::endl;
//     cutlass::TensorView<int32_t, cutlass::layout::RowMajor> view_ellIdx = tensor_ellIdx.host_view();
//     std::cout << view_ellIdx << std::endl;
//     std::cout << std::endl;


    cutlass::HostTensor<DataT_Cutlass, cutlass::layout::ColumnMajor> tensorB({B_rows, B_cols});
    for (i = 0; i < (B_rows); ++i) {
        for (j = 0; j < (B_cols); ++j) {
            // Write the element at location {i, j} in host memory
            tensorB.host_ref().at({i, j}) = (DataT_Cutlass)B_vals[i*B_cols + j];
        }
    }
    // Copy host memory to device memory
    tensorB.sync_device();
    // Obtain a device pointer usable in CUDA kernels
    DataT_Cutlass *device_ptrB = tensorB.device_data();

//     printf("Inside of %s\n", __func__);
//     std::cout << "view of tensorB:" << std::endl;
//     cutlass::TensorView<DataT_C, cutlass::layout::ColumnMajor> viewB = tensorB.host_view();
//     std::cout << viewB << std::endl;
//     std::cout << std::endl;


    cutlass::HostTensor<DataT_Cutlass, cutlass::layout::ColumnMajor> tensorC({rows, B_cols});
    for (i = 0; i < (rows); ++i) {
        for (j = 0; j < (B_cols); ++j) {
            // Write the element at location {i, j} in host memory
            tensorC.host_ref().at({i, j}) = (DataT_Cutlass) 0.0;
        }
    }
    // Copy host memory to device memory
    tensorC.sync_device();
    // Obtain a device pointer usable in CUDA kernels
    DataT_Cutlass *device_ptrC = tensorC.device_data();

    // ================================================================================

    hipDeviceProp_t props;

    hipError_t error = hipGetDeviceProperties(&props, 0);
    if (error != hipSuccess) {
        std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
        return -1;
    }

    if (__CUDACC_VER_MAJOR__ < 11 || props.major < 8) {

        //
        // This example requires an NVIDIA Ampere-architecture GPU.
        //

        std::cout
        << "CUTLASS's BlockedEll SpMM example requires a GPU of NVIDIA's Ampere Architecture or "
        << "later (compute capability 80 or greater).\n";
    }


    //
    // Define the BlockedEll type
    //

    using Gemm = typename cutlass::gemm::device::EllGemm<
        DataT_Cutlass,
        cutlass::layout::RowMajor,
        DataT_Cutlass,
        cutlass::layout::ColumnMajor,
        DataT_Cutlass,
        cutlass::layout::ColumnMajor,
        float,
        cutlass::arch::OpClassTensorOp,
        cutlass::arch::Sm80>;
    Gemm gemm_op;
    cutlass::Status status;

// -------------------------------------------------------------------------------------------------

    // Configure the GEMM arguments
    float alpha=1.0, beta=1.0;

    DataT_Cutlass *ptrD = tensorC.device_data();

    int lda = tensor.device_ref().stride(0);
    int ldb = tensorB.device_ref().stride(0);
    int ldc = tensorC.device_ref().stride(0);
    int ldd = tensorC.device_ref().stride(0);

    //initialize cuda events
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));

    // Configure GEMM arguments
    status = gemm_op({
      {rows, B_cols, cols},
      {tensor.device_ref(), lda},
      {tensorB.device_ref(), ldb},
      {tensorC.device_ref(), ldc},
      {ptrD, ldd},
      tensor_ellIdx.device_data(),
      ellValue_cols,
      ell_blocksize,
      0 /*options.a_base*/,
      {alpha, beta}
    });

    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize CUTLASS BlockedEll SpMM kernel." << std::endl;
      return(__LINE__);
    }

    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to run CUTLASS BlockedEll SpMM kernel." << std::endl;
      return(__LINE__);
    }

    // Wait for completion
    error = hipDeviceSynchronize();
    checkCudaErrors( hipEventRecord(stop, 0) );
    checkCudaErrors( hipEventSynchronize(stop) );
    checkCudaErrors( hipEventElapsedTime(&dt, start, stop) );
    checkCudaErrors( hipEventDestroy(start) );
    checkCudaErrors( hipEventDestroy(stop) );

    if (error != hipSuccess)  {
      std::cerr << "Kernel execution error: " << hipGetErrorString(error);
      return(__LINE__);
    }

    // ================================================================================

    tensorC.sync_host();
//     printf("Inside of %s\n", __func__);
//     std::cout << "view of tensorC:" << std::endl;
//     cutlass::TensorView<DataT_C, cutlass::layout::ColumnMajor> viewC = tensorC.host_view();
//     std::cout << viewC << std::endl;
//     std::cout << std::endl;
    for (i = 0; i < C_rows; ++i) {
        for (j = 0; j < C_cols; ++j) {
            // Write the element at location {i, j} in host memory
            C_vals[i*C_cols + j] = (T)tensorC.host_ref().at({i, j});
        }
    }

    return(0);

}

void bellpack_cutlass_multiplyAB(VBR* A, DataT* B, int B_cols, DataT_C* C, int C_cols, float& dt, int verbose) {

        // ellValue_cols, int *ell_blocksize, int *ellColInd_rows, int *ellColInd_cols, int *num_blocks, intT** ellColInd, Cutlass_DataT** ellValues
        int ell_blocksize, ellColInd_rows, ellColInd_cols, ellValue_cols, num_blocks;
        intT* ellColInd;
        DataT* ellValues;
        prepare_cusparse_BLOCKEDELLPACK(A, &ell_blocksize, &ellValue_cols, &ellColInd_rows, &ellColInd_cols, &num_blocks, &ellColInd, &ellValues);

        if (verbose > 1) {
            int pad_num = 0;
            for (int i=0; i<ellColInd_rows*ellColInd_cols; i++)
                if (ellColInd[i] == -1)
                    pad_num++;
            printf("ell_blocksize = %d, ellColInd has dimensions %d x %d with %d padding blocks, ellValues has dimensions %ld x %d\n", ell_blocksize, ellColInd_rows, ellColInd_cols, pad_num, A->rows, ellValue_cols);
        }

        compute_cutlass_bellpack<intT,DataT>(A->rows, A->cols, ell_blocksize, ellValue_cols, ellColInd, ellValues, A->cols, B_cols, B, A->rows, B_cols, C, dt);

        free(ellColInd);
        free(ellValues);

    return;
}

int cutlass_dense_multiplyAB(int m, int k, DataT* inputA, int n, DataT* inputB, float alp, float bet, DataT_C* output, float& dt) {

  dt = -1.0;

  // Define the GEMM operation
  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,                           // ElementA
    cutlass::layout::ColumnMajor,              // LayoutA
    cutlass::half_t,                           // ElementB
    cutlass::layout::ColumnMajor,              // LayoutB
    cutlass::half_t,                           // ElementOutput
    cutlass::layout::ColumnMajor,              // LayoutOutput
    float,                                     // ElementAccumulator
    cutlass::arch::OpClassTensorOp,            // tag indicating Tensor Cores
    cutlass::arch::Sm80                        // tag indicating target GPU compute architecture
  >;

  Gemm gemm_op;
  cutlass::Status status;

  //
  // Define the problem size
  //
  int M = m;
  int N = n;
  int K = k;

  float alpha = alp;
  float beta  = bet;

  //
  // Allocate device memory
  //



  cutlass::HostTensor<cutlass::half_t, cutlass::layout::ColumnMajor> A({M, K});
  cutlass::HostTensor<cutlass::half_t, cutlass::layout::ColumnMajor> B({K, N});
  cutlass::HostTensor<cutlass::half_t, cutlass::layout::ColumnMajor> C({M, N});

  // Input matrices to cutlass' structures



  for (int i=0; i<M; ++i)
    for (int j=0; j<K; ++j)
      A.host_ref().at({i, j}) = inputA[i*K +j];
  // Copy host memory to device memory
  A.sync_device();



  for (int i=0; i<K; ++i)
    for (int j=0; j<N; ++j)
      B.host_ref().at({i, j}) = inputB[i*N +j];
  // Copy host memory to device memory
  B.sync_device();



  cutlass::half_t const *ptrA = A.device_data();
  cutlass::half_t const *ptrB = B.device_data();
  cutlass::half_t const *ptrC = C.device_data();
  cutlass::half_t       *ptrD = C.device_data();



  int lda = A.device_ref().stride(0);
  int ldb = B.device_ref().stride(0);
  int ldc = C.device_ref().stride(0);
  int ldd = C.device_ref().stride(0);

  //
  // Launch GEMM on the device
  //



  status = gemm_op({
    {M, N, K},
    {ptrA, lda},            // TensorRef to A device tensor
    {ptrB, ldb},            // TensorRef to B device tensor
    {ptrC, ldc},            // TensorRef to C device tensor
    {ptrD, ldd},            // TensorRef to D device tensor - may be the same as C
    {alpha, beta}           // epilogue operation arguments
  });



  if (status != cutlass::Status::kSuccess) {
    return -1;
  }



  // Copy host memory to device memory
  C.sync_host();
  for (int i=0; i<M; ++i)
    for (int j=0; j<N; ++j)
      output[i*N +j] = C.host_ref().at({i, j});



  return 0;
}

// void cutlass_dense_multiplyAB(int rows, int cols, DataT* A, int B_rows, int B_cols, DataT* B, DataT_C* C, float& dt) {
//     int i, j;
//     cutlass::HostTensor<DataT_Cutlass, cutlass::layout::ColumnMajor> tensor({rows, cols});
//     for (i = 0; i < rows; ++i) {
//         for (j = 0; j < cols; ++j) {
//
//             // Write the element at location {i, j} in host memory
//             tensor.host_ref().at({i, j}) = (DataT_Cutlass)A[i*cols + j];
//
//         }
//     }
//
//     // Copy host memory to device memory
//     tensor.sync_device();
//
//     // Obtain a device pointer usable in CUDA kernels
//     DataT_Cutlass *device_ptr = tensor.device_data();
//
// //     std::cout << "view of tensor:" << std::endl;
// //     cutlass::TensorView<DataT_Cutlass, cutlass::layout::ColumnMajor> view = tensor.host_view();
// //     std::cout << view << std::endl;
// //     std::cout << std::endl;
//
//
//     cutlass::HostTensor<DataT_Cutlass, cutlass::layout::ColumnMajor> tensorB({B_rows, B_cols});
//
//
//     for (i = 0; i < (B_rows); ++i) {
//         for (j = 0; j < (B_cols); ++j) {
//
//             // Write the element at location {i, j} in host memory
//             tensorB.host_ref().at({i, j}) = B[i*B_cols + j];
//
//         }
//     }
//
//     // Copy host memory to device memory
//     tensorB.sync_device();
//
//     // Obtain a device pointer usable in CUDA kernels
//     DataT_Cutlass *device_ptrB = tensorB.device_data();
//
// //     std::cout << "view of tensorB:" << std::endl;
// //     cutlass::TensorView<DataT_Cutlass, cutlass::layout::ColumnMajor> viewB = tensorB.host_view();
// //     std::cout << viewB << std::endl;
// //     std::cout << std::endl;
//
//     cutlass::HostTensor<DataT_Cutlass, cutlass::layout::ColumnMajor> tensorC({rows, B_cols});
//
//
//     for (i = 0; i < (rows); ++i) {
//         for (j = 0; j < (B_cols); ++j) {
//
//             // Write the element at location {i, j} in host memory
//             tensorC.host_ref().at({i, j}) = (DataT_Cutlass) 0.0;
//
//         }
//     }
//
//     // Copy host memory to device memory
//     tensorC.sync_device();
//
//     // Obtain a device pointer usable in CUDA kernels
//     DataT_Cutlass *device_ptrC = tensorC.device_data();
//
// //     std::cout << "view of tensorC:" << std::endl;
// //     cutlass::TensorView<DataT_Cutlass, cutlass::layout::ColumnMajor> viewC = tensorC.host_view();
// //     std::cout << viewC << std::endl;
// //     std::cout << std::endl;
//
//   // ------------------------------------------------------------------------------
//
//   using Gemm = cutlass::gemm::device::Gemm<
//     DataT_Cutlass,                           // ElementA
//     cutlass::layout::ColumnMajor,              // LayoutA
//     DataT_Cutlass,                           // ElementB
//     cutlass::layout::ColumnMajor,              // LayoutB
//     DataT_Cutlass,                           // ElementOutput
//     cutlass::layout::ColumnMajor,              // LayoutOutput
//     float,                                     // ElementAccumulator
//     cutlass::arch::OpClassTensorOp,            // tag indicating Tensor Cores
//     cutlass::arch::Sm80                        // tag indicating target GPU compute architecture
//   >;
//
//   Gemm gemm_op_dn;
//   cutlass::Status status;
//
//   //
//   // Launch GEMM on the device
//   //
//
//   float alpha=1.0, beta=1.0;
//   int m = rows, k = cols, n = B_cols;
//
//   DataT_Cutlass *ptrD = tensorC.device_data();
//
//   int lda = tensor.device_ref().stride(0);
//   int ldb = tensorB.device_ref().stride(0);
//   int ldc = tensorC.device_ref().stride(0);
//   int ldd = tensorC.device_ref().stride(0);
//
//   //initialize cuda events
//   hipEvent_t start, stop;
//   checkCudaErrors(hipEventCreate(&start));
//   checkCudaErrors(hipEventCreate(&stop));
//   checkCudaErrors(hipEventRecord(start, 0));
//
//   status = gemm_op_dn({
//     {m, n, k},
//     {device_ptr, lda},
//     {device_ptrB, ldb},
//     {device_ptrC, ldc},
//     {ptrD, ldd},
//     {alpha, beta}
//   });
//
//   if (status != cutlass::Status::kSuccess) {
//     fprintf(stderr, "ERROR in file %s at line %d\n", __FILE__, __LINE__);
//   } else {
//     printf("SUCCESS in file %s at line %d\n", __FILE__, __LINE__);
//   }
//
//   // Wait for completion
//   checkCudaErrors( hipDeviceSynchronize() );
//   checkCudaErrors( hipEventRecord(stop, 0) );
//   checkCudaErrors( hipEventSynchronize(stop) );
//   checkCudaErrors( hipEventElapsedTime(&dt, start, stop) );
//   checkCudaErrors( hipEventDestroy(start) );
//   checkCudaErrors( hipEventDestroy(stop) );
//
//   tensorC.sync_host();
// //   std::cout << "view of tensorC:" << std::endl;
// //   std::cout << viewC << std::endl;
// //   std::cout << std::endl;
//
//   for (i = 0; i < (rows); ++i) {
//         for (j = 0; j < (B_cols); ++j) {
//
//             // Write the element at location {i, j} in host memory
//             C[i * B_cols + j] = (DataT_C)tensorC.host_ref().at({i, j});
//
//         }
//     }
//
//   return;
// }
